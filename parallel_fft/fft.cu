#include "hip/hip_runtime.h"
#include "fft.h"

thrust::complex<double> * dev_isamples;
thrust::complex<double> * dev_osamples;

void fft_init(int N)
{
	hipMalloc((void **)&dev_isamples, N * sizeof(int));
	checkCUDAError("hipMalloc dev_isamples failed!");
	hipMalloc((void **)&dev_osamples, N * sizeof(int));
	checkCUDAError("hipMalloc dev_osamples failed!");
}

void fft_free()
{
	hipFree(dev_isamples);
	hipFree(dev_osamples);
}

void ping_pong(thrust::complex ** a, thrust::complex ** b )
{
	thrust::complex * temp = * a;
	*a = *b;
	*b = temp;
}

/*
returns the reverse-bit value, normalized for original bit count
based on http://aggregate.org/MAGIC/#Bit%20Reversal
Assumes 32 bit int system
*/
__device__ int twiddle (unsigned int x)
{
    x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
    x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
    x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
    x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
    return ((x >> 16) | (x << 16)) >> (32 - ilog2ceil(x));
}

__global__ void inputScramble (int N, thrust::complex<double> * idata, thrust::complex<double> * odata)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index > N)
		return;

	//do global memory access
	thrust::complex<double> myVal = idata[index];

	//hide latency with computation
	int out_index = twiddle(index);

	odata[out_index] = myVal;
}

__global__ void doButterfly (int N, int stage, thrust::complex W, 
							thrust::complex * idata, thrust::complex * odata)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index > N)
		return;

	thrust::complex point = idata[index];

	// # points in this DFT computation
	float dft_points = powf(2, stage + 1); //logical shift instead?

	// N/2
	int half_points = (int) dft_points / 2; //also shift?

	// Relative index in this fourier transform
	int relativeIndex = index % dft_points;

	//GABE: What about multiiplicative factors??
	if (relativeIndex < half_points)
	{
		//subtract index
		thrust::complex point2 = idata[index+half_points];
		point = point + point2;
	}
	else
	{
		//subtract W^exp * index
		thrust::complex point2 = idata[index-half_points];
		float exponent = (relativeIndex % half_points) * (ilog2ceil(N) - stage);
		point = point2 - pow(W,exponent) * point;
	}

	odata[index] = point;
}

/*
parallel FFT implementation

inputs:
int N              - number of samples
float * samples    - pointer to array of sammples (of size N)
float * transform  - pointer to array where transform should be stored. 
                     It is safe for this to be the same as samples (i.e. in place)

*/

void parallel_fft (int N, 
	thrust::complex<double> * samples, 
	thrust::complex<double> * transform)
{
	//allocate buffers
	fft_init();

	//compute numBlocks
	dim3 numBlocks = (N + blockSize - 1) / blockSize;

	hipMemcpy(dev_isamples, samples, sizeof(thrust::complex<double>) * N, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy sample data to device failed!");

	//scrable inputs to reverse-binary order
	inputScramble << <numBlocks, blockSize>> >(N, dev_isamples, dev_osamples); 
	checkCUDAError("kernel inputScramble failed!");

	//ping pong buffers
	ping_pong(&dev_isamples, &dev_osamples);

	thrust::complex<double> W (thrust::cos(TWOPI / N), thrust::sin(TWOPI / N));

	//Butterfly
	for (int i = 0; i < ilog2ceil(N); ++i)
	{
		doButterfly(N, i, W, dev_isamples, dev_osamples);
		ping_pong(&dev_isamples, &dev_osamples);
	}

	//copy result to output
	hipMemcpy(transform, dev_isamples, N * sizeof(thrust::complex), hipMemcpyDeviceToHost);

	//free buffers 
	fft_free();

}