#include "hip/hip_runtime.h"
#include "fft.h"

#define blockSize 128
#define CHECKPOINT 1
thrust::complex<double> * dev_isamples;
thrust::complex<double> * dev_osamples;

#if CHECKPOINT
void checkpoint(const char * print_me, int N, thrust::complex<double> * buf)
{
	printf(print_me);

	for (int i = 0; i < N; ++i)
		printf("%f\n", thrust::abs(buf[i]));

}
#endif

__host__ __device__ int ilog2_2(int x) {
	int lg = 0;
	while (x >>= 1) {
		++lg;
	}
	return lg;
}

__host__ __device__ int ilog2ceil_2(int x) {
	return ilog2_2(x - 1) + 1;
}

void fft_init(int N)
{
	hipMalloc((void **)&dev_isamples, N * sizeof(thrust::complex<double>));
	checkCUDAError("hipMalloc dev_isamples failed!");
	hipMalloc((void **)&dev_osamples, N * sizeof(thrust::complex<double>));
	checkCUDAError("hipMalloc dev_osamples failed!");
}

void fft_free()
{
	hipFree(dev_isamples);
	hipFree(dev_osamples);
}

void ping_pong(thrust::complex<double> ** a, thrust::complex<double> ** b)
{
	thrust::complex<double> * temp = *a;
	*a = *b;
	*b = temp;
}




__device__ unsigned int twiddle(unsigned int x)
{
	//strictly reverses bits. must shift shift in calling context
	x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
	x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
	x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
	x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
	return ((x >> 16) | (x << 16));
}




__global__ void inputScramble(int N, thrust::complex<double> * idata, thrust::complex<double> * odata)
{
	
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (index >= N)
		return;

	//do global memory access
	thrust::complex<double> myVal = idata[index];

	//hide latency with computation
	int out_index = twiddle(index) >> (32 - ilog2ceil_2(N));
#if CHECKPOINT
	printf("iindex is %d oindex is %d\n", index, out_index, thrust::abs(myVal));
#endif
	odata[out_index] = myVal;
}




__global__ void doButterfly(int N, int stage, int numPoints,
	thrust::complex<double> * idata, thrust::complex<double> * odata)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= N)
		return;

	thrust::complex<double> point = idata[index];

	// N/2
	int half_points = numPoints / 2; //also shift?

	// Relative index in this fourier transform
	int relativeIndex = index % numPoints;

	thrust::complex<double> point2;

	if (relativeIndex < half_points)
	{
		// add point + N/2 to self
		point2 = idata[index + half_points];
	}
	else
	{
		// subtract self from - N/2
		point2 = idata[index - half_points];
		point *= -1.0;
		//thrust::complex<double> exponent = (relativeIndex % half_points) * (ilog2ceil_2(N) - stage);
		//point = point2 - thrust::pow(W, exponent) * point;
	}

	point = point + point2;

#if CHECKPOINT
	printf("i am %d, combining with %d\n", index, relativeIndex < half_points ? index + half_points : index - half_points);
	printf("half_points is %d, relativeIndex is %d\n", half_points, relativeIndex);
#endif

	odata[index] = point;
}






// in place multiplication of twiddle factors
__global__ void doMultiply(int N, int numPoints, thrust::complex<double> W, thrust::complex<double> * idata)
{
	
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (index >= N)
		return;

	//do global memory access
	thrust::complex<double> myVal = idata[index];

	int relativeIndex = index % numPoints;

	if (relativeIndex < numPoints / 2)
		return;

#if CHECKPOINT
	printf("my index is %d, myVal is %f + i%f, my exponent is %d\n", index, myVal.real(), myVal.imag() ,relativeIndex - numPoints / 2);
#endif

	thrust::complex<double> exponent = (relativeIndex - numPoints / 2, 0);
	myVal *= thrust::pow(W, exponent);

#if CHECKPOINT
	printf("my index is %d, newVal is %f + i%f\n", index, myVal.real(), myVal.imag());
#endif

	idata[index] = myVal;
}

/*
parallel FFT implementation

inputs:
int N              - number of samples
float * samples    - pointer to array of sammples (of size N)
float * transform  - pointer to array where transform should be stored. 
                     It is safe for this to be the same as samples (i.e. in place)

*/

void parallel_fft (int N, 
	thrust::complex<double> * samples, 
	thrust::complex<double> * transform)
{

#if CHECKPOINT
	checkpoint("initial samples\n", N, samples);
#endif

	//allocate buffers
	fft_init(N);

#if CHECKPOINT
	thrust::complex<double> * checkpoint_buf = (thrust::complex<double> *) calloc(N,sizeof(thrust::complex<double>));
#endif

	//compute numBlocks
	dim3 numBlocks = (N + blockSize - 1) / blockSize;

	hipMemcpy(dev_isamples, samples, sizeof(thrust::complex<double>) * N, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy sample data to device failed!");

#if CHECKPOINT
	hipMemcpy(checkpoint_buf, dev_isamples, N*sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
	checkpoint("initial samples on device\n", N, checkpoint_buf);
#endif


	//scrable inputs to reverse-binary order
	inputScramble << <numBlocks, blockSize>> >(N, dev_isamples, dev_osamples); 
	checkCUDAError("kernel inputScramble failed!");

#if CHECKPOINT
	hipMemcpy(checkpoint_buf, dev_osamples, N*sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
	checkpoint("after scramble\n", N, checkpoint_buf);
#endif

	//ping pong buffers
	ping_pong(&dev_isamples, &dev_osamples);

	//Butterfly
	for (int i = 0; i < ilog2ceil(N); ++i)
	{
		int numPoints = pow(2, i+1);
		// create the W vector for this N
		thrust::complex<double> W (cos((2.0 * M_PI) / numPoints),  -1.0 * sin((2.0 * M_PI) / numPoints));
		
		//pre-multiply pionts by necessary twiddle factors
		doMultiply << <numBlocks, blockSize>> >(N, numPoints, W, dev_isamples);
		checkCUDAError("doMultiply failed!");

		doButterfly << <numBlocks, blockSize>> >(N, i, numPoints, dev_isamples, dev_osamples);
		checkCUDAError("doButterfly sample data to device failed!");
		
#if CHECKPOINT
		hipMemcpy(checkpoint_buf, dev_osamples, N*sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
		checkpoint("after butterfly\n", N, checkpoint_buf);
#endif

		ping_pong(&dev_isamples, &dev_osamples);
	}

	//copy result to output
	hipMemcpy(transform, dev_isamples, N * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

	//free buffers 
	fft_free();

#if CHECKPOINT
	free(checkpoint_buf);
#endif
}
