#include "hip/hip_runtime.h"
#include "fft.h"

#define blockSize 128

thrust::complex<double> * dev_isamples;
thrust::complex<double> * dev_osamples;

__host__ __device__ int ilog2_2(int x) {
	int lg = 0;
	while (x >>= 1) {
		++lg;
	}
	return lg;
}

__host__ __device__ int ilog2ceil_2(int x) {
	return ilog2_2(x - 1) + 1;
}

void fft_init(int N)
{
	hipMalloc((void **)&dev_isamples, N * sizeof(int));
	checkCUDAError("hipMalloc dev_isamples failed!");
	hipMalloc((void **)&dev_osamples, N * sizeof(int));
	checkCUDAError("hipMalloc dev_osamples failed!");
}

void fft_free()
{
	hipFree(dev_isamples);
	hipFree(dev_osamples);
}

void ping_pong(thrust::complex<double> ** a, thrust::complex<double> ** b)
{
	thrust::complex<double> * temp = *a;
	*a = *b;
	*b = temp;
}


__device__ int twiddle(unsigned int x)
{
	x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
	x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
	x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
	x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
	return ((x >> 16) | (x << 16)) >> (32 - ilog2ceil_2(x));
}

__global__ void inputScramble(int N, thrust::complex<double> * idata, thrust::complex<double> * odata)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index > N)
		return;

	//do global memory access
	thrust::complex<double> myVal = idata[index];

	//hide latency with computation
	int out_index = twiddle(index);

	odata[out_index] = myVal;
}

__global__ void doButterfly(int N, int stage, thrust::complex<double> W,
	thrust::complex<double> * idata, thrust::complex<double> * odata)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index > N)
		return;

	thrust::complex<double> point = idata[index];

	// # points in this DFT computation
	int dft_points = (int)powf(2, stage + 1); //logical shift instead?

	// N/2
	int half_points = (int)dft_points / 2; //also shift?

	// Relative index in this fourier transform
	int relativeIndex = index % dft_points;

	//GABE: What about multiiplicative factors??
	if (relativeIndex < half_points)
	{
		//subtract index
		thrust::complex<double> point2 = idata[index + half_points];
		point = point + point2;
	}
	else
	{
		//subtract W^exp * index
		thrust::complex<double> point2 = idata[index - half_points];
		thrust::complex<double> exponent = (relativeIndex % half_points) * (ilog2ceil_2(N) - stage);
		point = point2 - thrust::pow(W, exponent) * point;
	}

	odata[index] = point;
}

/*
parallel FFT implementation

inputs:
int N              - number of samples
float * samples    - pointer to array of sammples (of size N)
float * transform  - pointer to array where transform should be stored. 
                     It is safe for this to be the same as samples (i.e. in place)

*/

void parallel_fft (int N, 
	thrust::complex<double> * samples, 
	thrust::complex<double> * transform)
{
	// Radix 2 FFT operates on Powers of Two. Pad as needed.
	//GABE pad here

	//allocate buffers
	fft_init(N);

	//compute numBlocks
	dim3 numBlocks = (N + blockSize - 1) / blockSize;

	hipMemcpy(dev_isamples, samples, sizeof(thrust::complex<double>) * N, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy sample data to device failed!");

	//scrable inputs to reverse-binary order
	inputScramble << <numBlocks, blockSize>> >(N, dev_isamples, dev_osamples); 
	checkCUDAError("kernel inputScramble failed!");

	//ping pong buffers
	ping_pong(&dev_isamples, &dev_osamples);

	thrust::complex<double> W (cos((2.0 * M_PI) / N), sin((2.0 * M_PI) / N));

	//Butterfly
	for (int i = 0; i < ilog2ceil(N); ++i)
	{
		doButterfly << <numBlocks, blockSize>> >(N, i, W, dev_isamples, dev_osamples);
		ping_pong(&dev_isamples, &dev_osamples);
	}

	//copy result to output
	hipMemcpy(transform, dev_isamples, N * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

	//free buffers 
	fft_free();

}
